#include "hip/hip_runtime.h"
#include "dali/pipeline/operators/reader/nvdecoder/imgproc.h"

#include <hip/hip_fp16.h>

namespace dali {

namespace {

// using math from https://msdn.microsoft.com/en-us/library/windows/desktop/dd206750(v=vs.85).aspx

template<typename T>
struct yuv {
    T y, u, v;
};

// https://docs.microsoft.com/en-gb/windows/desktop/medfound/recommended-8-bit-yuv-formats-for-video-rendering#converting-8-bit-yuv-to-rgb888
__constant__ float yuv2rgb_mat_norm[9] = {
    1.164383f,  0.0f,       1.596027f,
    1.164383f, -0.391762f, -0.812968f,
    1.164383f,  2.017232f,  0.0f
};

// not normalized need *255
__constant__ float yuv2rgb_mat[9] = {
    1.164383f * 255.f,  0.0f,       1.596027f * 255.f,
    1.164383f * 255.f, -0.391762f * 255.f, -0.812968f * 255.f,
    1.164383f * 255.f,  2.017232f * 255.f,  0.0f
};

__device__ float clip(float x, float max) {
    return fminf(fmaxf(x, 0.0f), max);
}

template<typename T>
__device__ T convert(const float x) {
    return static_cast<T>(x);
}

template<>
__device__ half convert<half>(const float x) {
    return __float2half(x);
}

template<>
__device__ uint8_t convert<uint8_t>(const float x) {
    return static_cast<uint8_t>(roundf(x));
}

template<typename YUV_T, typename RGB_T, bool Normalized = false>
__device__ void yuv2rgb(const yuv<YUV_T>& yuv, RGB_T* rgb,
                        size_t stride) {
    auto y = (static_cast<float>(yuv.y) - 16.0f/255);
    auto u = (static_cast<float>(yuv.u) - 128.0f/255);
    auto v = (static_cast<float>(yuv.v) - 128.0f/255);


    float r, g, b;
    if (Normalized) {
        auto& m = yuv2rgb_mat_norm;
        r = clip(y*m[0] + u*m[1] + v*m[2], 1.0);
        g = clip(y*m[3] + u*m[4] + v*m[5], 1.0);
        b = clip(y*m[6] + u*m[7] + v*m[8], 1.0);
    } else {
        auto& m = yuv2rgb_mat;
        r = clip(y*m[0] + u*m[1] + v*m[2], 255.0);
        g = clip(y*m[3] + u*m[4] + v*m[5], 255.0);
        b = clip(y*m[6] + u*m[7] + v*m[8], 255.0);
    }

    rgb[0] = convert<RGB_T>(r);
    rgb[stride] = convert<RGB_T>(g);
    rgb[stride*2] = convert<RGB_T>(b);
}

template<typename T>
__global__ void process_frame_kernel(
    hipTextureObject_t luma, hipTextureObject_t chroma,
    T* dst, int index,
    float fx, float fy,
    int dst_width, int dst_height, int c) {

    const int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (dst_x >= dst_width || dst_y >= dst_height)
        return;

    auto src_x = 0.0f;
    src_x = static_cast<float>(dst_x) * fx;
    auto src_y = static_cast<float>(dst_y) * fy;


    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#tex2d-object
    yuv<float> yuv;
    yuv.y = tex2D<float>(luma, src_x + 0.5, src_y + 0.5);
    auto uv = tex2D<float2>(chroma, (src_x / 2) + 0.5, (src_y / 2) + 0.5);
    yuv.u = uv.x;
    yuv.v = uv.y;

    auto* out = &dst[(dst_x + dst_y * dst_width) * c];

    size_t stride = 1;
    // TODO(spanev) Handle normalized version
    yuv2rgb<float, float, false>(yuv, out, stride);
}

inline constexpr int divUp(int total, int grain) {
    return (total + grain - 1) / grain;
}

} //  namespace

template<typename T>
void process_frame(
    hipTextureObject_t chroma, hipTextureObject_t luma,
    SequenceWrapper& output, int index, hipStream_t stream,
    uint16_t input_width, uint16_t input_height) {
    auto scale_width = input_width;
    auto scale_height = input_height;

    auto fx = static_cast<float>(input_width) / scale_width;
    auto fy = static_cast<float>(input_height) / scale_height;

    dim3 block(32, 8);
    dim3 grid(divUp(output.width, block.x), divUp(output.height, block.y));

    int frame_stride = index * output.height * output.width * output.channels;
    LOG_LINE << "Processing frame " << index << " (frame_stride=" << frame_stride << ")" << std::endl;
    auto* tensor_out = output.sequence.mutable_data<T>() + frame_stride;

    process_frame_kernel<<<grid, block, 0, stream>>>
            (luma, chroma, tensor_out, index, fx, fy, output.width, output.height, output.channels);
}

template
void process_frame<float>(
    hipTextureObject_t chroma, hipTextureObject_t luma,
    SequenceWrapper& output, int index, hipStream_t stream,
    uint16_t input_width, uint16_t input_height);

}  // namespace dali