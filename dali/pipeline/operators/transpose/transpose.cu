// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "dali/pipeline/operators/transpose/transpose.h"
#include "dali/error_handling.h"

namespace dali {

#define cuttCheck(stmt) do {                                   \
  cuttResult err = stmt;                                       \
  if (err != CUTT_SUCCESS) {                                   \
    DALI_FAIL("Error while transposing" + std::string(#stmt)); \
  }                                                            \
} while(0)

template <>
Transpose<GPUBackend>::~Transpose() {
  if (cutt_handle_ > 0) {
    cuttCheck(cuttDestroy(cutt_handle_));
  }
}


template <>
void Transpose<GPUBackend>::NaiveTransposeKernel(const TensorList<GPUBackend>& input,
                                     TensorList<GPUBackend>* output) {

}

template <>
template <typename T>
void Transpose<GPUBackend>::cuTTKernel(const TensorList<GPUBackend>& input,
                           TensorList<GPUBackend>* output,
                           hipStream_t stream) {
  Dims tmp = input.tensor_shape(0);
  std::vector<int> input_shape(tmp.begin(), tmp.end());

  int *dim = const_cast<int*>(input_shape.data());
  int *permutation = const_cast<int*>(perm_.data());

  if (cutt_handle_ == 0) {
    cuttCheck(cuttPlan(&cutt_handle_, perm_.size(), dim, permutation, sizeof(T), stream));
  }

  for (int i = 0; i < batch_size_; ++i) {
    const void* in = input.raw_tensor(i);
    void* out = output->raw_mutable_tensor(i);
    cuttCheck(cuttExecute(cutt_handle_, in, out));
  }
}


template<>
void Transpose<GPUBackend>::SetupSharedSampleParams(DeviceWorkspace *ws) {
  auto &input = ws->Input<GPUBackend>(0);
  auto* tl_sequence_output = ws->Output<GPUBackend>(0);
  tl_sequence_output->set_type(TypeInfo::Create<float>());
}


inline Dims GetPermutedDims(const Dims& dims, const std::vector<int>& permutation) {
  Dims permuted_dims;
  for (auto idx : permutation) {
    permuted_dims.push_back(dims[idx]);
  }
  return permuted_dims;
}

template<>
void Transpose<GPUBackend>::RunImpl(DeviceWorkspace* ws, int idx) {
  const auto& input = ws->Input<GPUBackend>(idx);
  auto* output = ws->Output<GPUBackend>(idx);

  Dims input_shape = input.tensor_shape(0);
  DALI_ENFORCE(input_shape.size() == perm_.size(),
               "Transposed tensors rank should be equal to the permutation index list.");

  if (input.IsDenseTensor()) {
    Dims permuted_dims = GetPermutedDims(input_shape, perm_);
    output->Resize(std::vector<Dims>(batch_size_, permuted_dims));
    cuTTKernel(input, output, ws->stream());
  } else {
    std::vector<Dims> tl_shape;
    for (int i = 0; i < batch_size_; ++i) {
      Dims in_shape = input.tensor_shape(i);
      tl_shape.emplace_back(GetPermutedDims(in_shape, perm_));
    }
    output->Resize(tl_shape);
    NaiveTransposeKernel(input, output);
  }
}

DALI_REGISTER_OPERATOR(Transpose, Transpose<GPUBackend>, GPU);

}  // namespace dali
